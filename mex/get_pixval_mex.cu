#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "tmwtypes.h"

// Block Size
static const int BS = 16;

__global__ void cuda_get_pp_by_rc (const int      K,
                                   const int      L,
                                   const int      ML,
                                   const int      N,
                                   const float    *p,
                                   const float    *rcc,
                                   const uint32_T *rci,
                                   float          *pp)
{
  //// make sure we're working within the range of pp

  // the index along dim1 of pp: 0 or 1
  int dim_pnt = threadIdx.z;

  // the index along dim2 of pp: ml
  int sub_ml    = threadIdx.x;
  int blkcnt_ml = blockIdx.x;
  int ml        = BS*blkcnt_ml + sub_ml;
  if (ml >= ML) return;

  // the index along dim3 of pp: n
  int sub_n    = threadIdx.y;
  int blkcnt_n = blockIdx.y;
  int n        = BS*blkcnt_n + sub_n;
  if (n >= N) return;

  
  //// do the job

  // initialize the pp value
  float val = 0.0;

  // weighted sum of the K points
  for (int k = 0; k < K; ++k) {
    // which point
    int i_pnt = *(rci + k + ml*K);
    i_pnt -= 1; // matlab 1-base -> C 0-base

    // the point value
    float pv = *(p + dim_pnt + i_pnt*2 + n*2*L);

    // what coefficient
    float w = *(rcc + k + ml*K);

    // the output point value
    val += pv*w;
  } // for k

  // write to the target
  *(pp + dim_pnt + ml*2 + n*2*ML) = val;

  __syncthreads();

}

__global__ void cuda_get_ind_val (const int   H,
                                  const int   W,
                                  const int   ML,
                                  const int   N,
                                  const float *I,
                                  const float *pp,
                                  float       *f,
                                  uint32_T    *ind )
{
  //// make sure we're working within the range of pp

  // the index along dim2 of pp: ml
  int sub_ml    = threadIdx.x;
  int blkcnt_ml = blockIdx.x;
  int ml        = BS*blkcnt_ml + sub_ml;
  if (ml >= ML) return;

  // the index along dim3 of pp: n
  int sub_n    = threadIdx.y;
  int blkcnt_n = blockIdx.y;
  int n        = BS*blkcnt_n + sub_n;
  if (n >= N) return;


  //// do the job

  // get (py, px) from pp [2, ML, N],
  int i_pntx = 0 + ml*2 + n*2*ML; // the convention of the pp storage: x first, y second
  int i_pnty = 1 + ml*2 + n*2*ML;
  // normalized coordinate -> integer coordinate
  int py = int( float(H) * pp[i_pnty] ); // [0,1] -> {0,1,...,H-1}
  int px = int( float(W) * pp[i_pntx] ); // [0,1] -> {0,1,...,W-1}
  py = (py<H) ? (py) : (H-1); // make it in the range
  px = (px<W) ? (px) : (W-1); 
  // convert it to linear index to the image I,
  // i.e., the linear index for (py,px,0,n) at I [H,W,3,N] 
  int i_pixval = py + px*H + n*H*W*3; // py + px*H + 0*W*H + n*H*W*3;


  //// fill the output
  int i_out = ml + ML*n; // the linear index for the output 
  f[i_out] = I[i_pixval];  // fill the feature f
  ind[i_out] = (uint32_T)(i_pixval + 1);  // fill the index ind: 0-base -> 1-base

  __syncthreads();

}


// [I,p,rcc,rci] = check_and_get_input(nin,in); Helper
void check_and_get_input (int              nin, 
                          mxArray    const *in[],
                          mxGPUArray const *&I, 
                          mxGPUArray const *&p,
                          mxGPUArray const *&rcc,
                          mxGPUArray const *&rci)
{
  if (nin != 4)
    mexErrMsgTxt("Incorrect arguments. [f,ind] = get_pixval(I, p, rcc, rci)");

  //// check if gpuArray
  if ( mxIsGPUArray( in[0] ) == 0 ) mexErrMsgTxt("I must be a gpuArray.");
  if ( mxIsGPUArray( in[1] ) == 0 ) mexErrMsgTxt("p must be a gpuArray.");
  if ( mxIsGPUArray( in[2] ) == 0 ) mexErrMsgTxt("rcc must be a gpuArray.");
  if ( mxIsGPUArray( in[3] ) == 0 ) mexErrMsgTxt("rci must be a gpuArray."); 
  
  //// fetch the results
  I   = mxGPUCreateFromMxArray( in[0] );
  p   = mxGPUCreateFromMxArray( in[1] );
  rcc = mxGPUCreateFromMxArray( in[2] );
  rci = mxGPUCreateFromMxArray( in[3] );

  //// check the types
  if (mxGPUGetClassID(I)   != mxSINGLE_CLASS ) mexErrMsgTxt("I must be the type single.");
  if (mxGPUGetClassID(p)   != mxSINGLE_CLASS ) mexErrMsgTxt("p must be the type single.");
  if (mxGPUGetClassID(rcc) != mxSINGLE_CLASS ) mexErrMsgTxt("rcc must be the type single.");
  if (mxGPUGetClassID(rci) != mxUINT32_CLASS ) mexErrMsgTxt("rci must be the type uint32.");
}

// pp = get_pp_by_rc(p,rcc,rci); Get all the points pp by random combination
void get_pp_by_rc (mxGPUArray const *p, 
                   mxGPUArray const *rcc, 
                   mxGPUArray const *rci,
                   mxGPUArray       *pp)
{
  //// raw pointer
  const float    *ptr_p   = (const float*)    ( mxGPUGetDataReadOnly(p) );
  const float    *ptr_rcc = (const float*)    ( mxGPUGetDataReadOnly(rcc) );
  const uint32_T *ptr_rci = (const uint32_T*) ( mxGPUGetDataReadOnly(rci) );
  float          *ptr_pp  = (float*)          ( mxGPUGetData(pp) );

  //// auxiliary 
  const int K  = *( 0 + mxGPUGetDimensions(rcc) ); // rcc [K, ML]
  const int ML = *( 1 + mxGPUGetDimensions(rcc) );
  const int L  = *( 1 + mxGPUGetDimensions(p)   ); // p [2,L,N]
  const int N  = *( 2 + mxGPUGetDimensions(p)   ); // p [2,L,N]

  //// block and thread partition
  dim3 num_thd( BS, BS, 2 );
  dim3 num_blk( (ML+BS-1)/BS, (N+BS-1)/BS );

#ifndef NDEBUG
  mexPrintf("In get_pp_by_rc\n");
  mexPrintf("K = %d\n", K);
  mexPrintf("ML = %d\n", ML);
  mexPrintf("L = %d\n", L);
  mexPrintf("N = %d\n", N);
#endif // !NDEBUG

  cuda_get_pp_by_rc<<<num_blk, num_thd>>>(K,L,ML,N, ptr_p,ptr_rcc,ptr_rci,  ptr_pp);
}

// [f,ind] = get_ind_val(I,pp); Get the values and the index 
void get_ind_val (mxGPUArray const *I, 
                  mxGPUArray const *pp,
                  mxGPUArray       *f, 
                  mxGPUArray       *ind)
{
  // thread 
  dim3 num_thd(BS,BS);
  // block
  const int ML = *( 1 + mxGPUGetDimensions(pp) ); // p [2,L,N]
  const int N  = *( 2 + mxGPUGetDimensions(pp) ); 
  dim3 num_blk( (ML+BS-1)/BS, (N+BS-1)/BS );
  // image size
  const int H = *( 0 + mxGPUGetDimensions(I) ); // I [H, W, 3, N]
  const int W = *( 1 + mxGPUGetDimensions(I) ); 
  // raw pointer
  const float    *ptr_I   = (const float*) ( mxGPUGetDataReadOnly(I) );
  const float    *ptr_pp  = (const float*) ( mxGPUGetDataReadOnly(pp) );
  float          *ptr_f   = (float*)       ( mxGPUGetData(f) );
  uint32_T       *ptr_ind = (uint32_T*)    ( mxGPUGetData(ind) );

#ifndef NDEBUG
  mexPrintf("In get_ind_val\n");
  mexPrintf("H = %d\n", H);
  mexPrintf("W = %d\n", W);
  mexPrintf("ML = %d\n", ML);
  mexPrintf("N = %d\n", N);
#endif // !NDEBUG

  cuda_get_ind_val<<<num_blk, num_thd>>>(H,W,ML,N, ptr_I,ptr_pp,   ptr_f,ptr_ind);
}



// [f,ind] = get_pixval(I, p, rcc, rci)
// f:   [MLN]     features
// ind: [MLN]     the linear index
// I:   [H,W,3,N] image array
// p:   [2,L,N]   points
// rcc: [K, ML]   combination coefficients
// rci: [K, ML]   non zero elements index
void mexFunction(int nout, mxArray *out[],
                 int nin,  mxArray const *in[])
{
  //// Prepare the Input
  mxGPUArray const *I;
  mxGPUArray const *p;
  mxGPUArray const *rcc;
  mxGPUArray const *rci;
  check_and_get_input(nin, in,  I,p,rcc,rci);


  //// Create the Output
  const mwSize *ddd = mxGPUGetDimensions(rcc);
  size_t ML         = *(ddd + 1);
  const mwSize *dim = mxGPUGetDimensions(p);
  mwSize N          = *(dim + 2); 
  mwSize dimo[1];
  dimo[0] = ML*N;
  mxGPUArray *f   = mxGPUCreateGPUArray(1, dimo, mxSINGLE_CLASS, mxREAL, // [MLN]
                                        MX_GPU_DO_NOT_INITIALIZE); 
  out[0]          = mxGPUCreateMxArrayOnGPU(f);
  mxGPUArray *ind = mxGPUCreateGPUArray(1, dimo, mxUINT32_CLASS, mxREAL, // [MLN]
                                        MX_GPU_DO_NOT_INITIALIZE); 
  out[1]          = mxGPUCreateMxArrayOnGPU(ind);


  //// do the job

  // get all the points pp: [2, ML, N]
  mwSize pp_dim[3];
  pp_dim[0] = 2;
  pp_dim[1] = ML;
  pp_dim[2] = N;
  mxGPUArray *pp = mxGPUCreateGPUArray (3, pp_dim, mxSINGLE_CLASS, mxREAL, // [MLN]
                                        MX_GPU_DO_NOT_INITIALIZE);
  get_pp_by_rc (p,rcc,rci,  pp);
  out[2] = mxGPUCreateMxArrayOnGPU(pp);

  // get the linear index and the values
  get_ind_val (I,pp, f,ind);

  // cleanup !!!
  mxGPUDestroyGPUArray(I);
  mxGPUDestroyGPUArray(p);
  mxGPUDestroyGPUArray(rcc);
  mxGPUDestroyGPUArray(rci);
  mxGPUDestroyGPUArray(pp);
  mxGPUDestroyGPUArray(f);
  mxGPUDestroyGPUArray(ind);

  return;
}
