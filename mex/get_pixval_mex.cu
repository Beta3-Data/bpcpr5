#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "tmwtypes.h"


void __global__ cuda_get_pp_by_rc (const int      K,
                                   const int      L,
                                   const float    *p,
                                   const float    *rcc,
                                   const uint32_T *rci,
                                   float          *pp)
{
  int ml = blockIdx.x;
  int ML = gridDim.x; // ???
  int i_pntdim = threadIdx.x;
  int n  = threadIdx.y;

  // initialize the pp value
  *(pp + i_pntdim + ml*2 + n*2*ML) = (float) 0.0;
  
  // accumulate the combination of the K points
  for (int k = 0; k < K; ++k) {
    *(pp + k) = (float) ( (k+1)*1.57 );

    // which point
    int i_pnt = *(rci + k + ml*K);
    i_pnt -= 1; // matlab 1-base -> C 0-base

    // the point value
    float pv = *(p + i_pntdim + i_pnt*2 + n*2*L);

    // what coefficient
    float w = *(rcc + k + ml*K);

    // the output point value
    *(pp + i_pntdim + ml*2 + n*2*ML) += pv*w;
    
  }

}

void __global__ cuda_get_ind_val (const float    *I,
                                  const float    *pp,
                                  const float    *f,
                                  const uint32_T *ind )
{

}


// [I,p,rcc,rci] = check_and_get_input(nin,in); Helper
void check_and_get_input (int              nin, 
                          mxArray    const *in[],
                          mxGPUArray const *&I, 
                          mxGPUArray const *&p,
                          mxGPUArray const *&rcc,
                          mxGPUArray const *&rci)
{
  if (nin != 4)
    mexErrMsgTxt("Incorrect arguments. [f,ind] = get_pixval(I, p, rcc, rci)");

  //// check if gpuArray
  if ( mxIsGPUArray( in[0] ) == 0 ) mexErrMsgTxt("I must be a gpuArray.");
  if ( mxIsGPUArray( in[1] ) == 0 ) mexErrMsgTxt("p must be a gpuArray.");
  if ( mxIsGPUArray( in[2] ) == 0 ) mexErrMsgTxt("rcc must be a gpuArray.");
  if ( mxIsGPUArray( in[3] ) == 0 ) mexErrMsgTxt("rci must be a gpuArray."); 
  
  //// fetch the results
  I   = mxGPUCreateFromMxArray( in[0] );
  p   = mxGPUCreateFromMxArray( in[1] );
  rcc = mxGPUCreateFromMxArray( in[2] );
  rci = mxGPUCreateFromMxArray( in[3] );

  //// check the types
  if (mxGPUGetClassID(I)   != mxSINGLE_CLASS ) mexErrMsgTxt("I must be the type single.");
  if (mxGPUGetClassID(p)   != mxSINGLE_CLASS ) mexErrMsgTxt("p must be the type single.");
  if (mxGPUGetClassID(rcc) != mxSINGLE_CLASS ) mexErrMsgTxt("rcc must be the type single.");
  if (mxGPUGetClassID(rci) != mxUINT32_CLASS ) mexErrMsgTxt("rci must be the type uint32.");
}

// pp = get_pp_by_rc(p,rcc,rci); Get all the points pp by random combination
void get_pp_by_rc (mxGPUArray const *p, 
                   mxGPUArray const *rcc, 
                   mxGPUArray const *rci,
                   mxGPUArray       *pp)
{
  const float    *ptr_p   = (const float*)    ( mxGPUGetDataReadOnly(p) );
  const float    *ptr_rcc = (const float*)    ( mxGPUGetDataReadOnly(rcc) );
  const uint32_T *ptr_rci = (const uint32_T*) ( mxGPUGetDataReadOnly(rci) );
  float          *ptr_pp  = (float*)          ( mxGPUGetData(pp) );

  const int K  = *( 0 + mxGPUGetDimensions(rcc) ); // rcc [K, ML]
  const int ML = *( 1 + mxGPUGetDimensions(rcc) );
  const int L  = *( 1 + mxGPUGetDimensions(p)   ); // p [2,L,N]
  const int N  = *( 2 + mxGPUGetDimensions(p)   ); // p [2,L,N]
  dim3 num_thd(2, N); // 2 dimensional point

#ifndef NDEBUG
  mexPrintf("In get_pp_by_rc\n");
  mexPrintf("K = %d\n", K);
  mexPrintf("ML = %d\n", ML);
  mexPrintf("L = %d\n", L);
  mexPrintf("N = %d\n", N);
#endif // !NDEBUG

  cuda_get_pp_by_rc<<<ML, num_thd>>>(K,L,ptr_p,ptr_rcc,ptr_rci,  ptr_pp);
}

// [f,ind] = get_ind_val(I,pp); Get the values and the index 
void get_ind_val (mxGPUArray const *I, 
                  mxGPUArray const *pp,
                  mxGPUArray const *f, 
                  mxGPUArray const *ind)
{
  const float    *ptr_I   = (const float*)    ( mxGPUGetDataReadOnly(I) );
  const float    *ptr_pp  = (const float*)    ( mxGPUGetDataReadOnly(pp) );
  const float    *ptr_f   = (const float*)    ( mxGPUGetDataReadOnly(f) );
  const uint32_T *ptr_ind = (const uint32_T*) ( mxGPUGetDataReadOnly(ind) );

  cuda_get_ind_val<<<1, 1>>>(ptr_I,ptr_pp,  ptr_f,ptr_ind);
}



// [f,ind] = get_pixval(I, p, rcc, rci)
// f:   [MLN]     features
// ind: [MLN]     the linear index
// I:   [H,W,3,N] image array
// p:   [2,L,N]   points
// rcc: [K, ML]   combination coefficients
// rci: [K, ML]   non zero elements index
void mexFunction(int nout, mxArray *out[],
                 int nin,  mxArray const *in[])
{
  //// Prepare the Input
  mxGPUArray const *I;
  mxGPUArray const *p;
  mxGPUArray const *rcc;
  mxGPUArray const *rci;
  check_and_get_input(nin, in,  I,p,rcc,rci);


  //// Create the Output
  const mwSize *ddd = mxGPUGetDimensions(rcc);
  size_t ML         = *(ddd + 1);
  const mwSize *dim = mxGPUGetDimensions(p);
  mwSize N          = *(dim + 2); 
  mwSize dimo[1];
  dimo[0] = ML*N;
  mxGPUArray *f   = mxGPUCreateGPUArray(1, dimo, mxSINGLE_CLASS, mxREAL, // [MLN]
                                        MX_GPU_DO_NOT_INITIALIZE); 
  out[0]          = mxGPUCreateMxArrayOnGPU(f);
  mxGPUArray *ind = mxGPUCreateGPUArray(1, dimo, mxUINT32_CLASS, mxREAL, // [MLN]
                                        MX_GPU_DO_NOT_INITIALIZE); 
  out[1]          = mxGPUCreateMxArrayOnGPU(ind);


  //// do the job

  // get all the points pp: [2, ML, N]
  mwSize pp_dim[3];
  pp_dim[0] = 2;
  pp_dim[1] = ML;
  pp_dim[2] = N;
  mxGPUArray *pp = mxGPUCreateGPUArray (3, pp_dim, mxSINGLE_CLASS, mxREAL, // [MLN]
                                        MX_GPU_DO_NOT_INITIALIZE);
  get_pp_by_rc (p,rcc,rci,  pp);
  out[2] = mxGPUCreateMxArrayOnGPU(pp);

  // get the linear index and the values
  get_ind_val (I,pp, f,ind);

  // cleanup !!!
  mxGPUDestroyGPUArray(I);
  mxGPUDestroyGPUArray(p);
  mxGPUDestroyGPUArray(rcc);
  mxGPUDestroyGPUArray(rci);
  mxGPUDestroyGPUArray(pp);
  mxGPUDestroyGPUArray(f);
  mxGPUDestroyGPUArray(ind);

  return;
}
